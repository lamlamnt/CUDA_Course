#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"

__global__ void do_work(double *data, int N, int idx, int chunk_num) {
	printf("Start doing computing %d \n", chunk_num);
	int i = blockIdx.x * blockDim.x + blockDim.x*idx + threadIdx.x;
	if (i < N) {
		for (int j = 0; j < 20; j++) {
			data[i] = cos(data[i]);
			data[i] = sqrt(fabs(data[i]));
		}
	}
}

int main()
{
	//Allocate 1 GB of data
	int total_data = 1<<27;
	//double *d_data;
	double *h_data;
	int stream_number = 3; //One stream that copies host to device, one that does computation, and one that copies device to host
	int num_chunks = 10;
	h_data = (double*)malloc(total_data*sizeof(double));
	//checkCudaErrors(hipMalloc( (void**)&d_data, total_data*sizeof(double) ));

	//Initialise host data
	srand(0);
	for (int i = 0; i < total_data; i++)
		h_data[i] = (double)rand()/(double)RAND_MAX;

	//Start timing	
	float time;
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	//Divide the host data up to multiple parts 
	double *h_data_list[num_chunks]; 
	double *d_data_list[num_chunks];
	//Assume it's divisible!!!
	int num_elements_per_chunk = (int)total_data/num_chunks;
	for (int i = 0; i < num_chunks; ++i) 
	{
        h_data_list[i] = &h_data[i*num_elements_per_chunk];
    }

	//Stream that copies the data from host to device
	hipStream_t stream_copyhd;
	hipStreamCreate(&stream_copyhd);
	//Use pinned memory on the host
	for(int i = 0; i < num_chunks; ++i)
	{
		//allocates fixed memory to the CPU
		printf("Start copying chunk of data %d from host to device \n",i);
		hipError_t status = hipHostMalloc((void**)&h_data_list[i], sizeof(double)*num_elements_per_chunk);
		if (status != hipSuccess)
  			printf("Error allocating pinned host memory\n");
		checkCudaErrors(hipMalloc( (void**)&d_data_list[i], num_elements_per_chunk*sizeof(double) ));
		hipMemcpyAsync(d_data_list[i],h_data_list[i],num_elements_per_chunk*sizeof(double),hipMemcpyHostToDevice,stream_copyhd);
	}

	//Stream that does the computation
	hipStream_t stream_compute;
	hipStreamCreate(&stream_compute);
	int blocksize_stream = 256;
	int nblocks_stream = (num_elements_per_chunk-1)/blocksize_stream + 1;
	for(int i = 0; i < num_chunks; ++i)
	{
		printf("Launch kernel for chunk %d \n",i);
		do_work<<<blocksize_stream,nblocks_stream,0,stream_compute>>>(d_data_list[i], num_elements_per_chunk, nblocks_stream, i);
	}

	hipStream_t stream_copydh;
	hipStreamCreate(&stream_copydh);
	for(int i = 0; i < num_chunks;++i)
	{
		printf("Start copying chunk of data %d from device to host \n",i);
		hipMemcpyAsync(h_data_list[i],d_data_list[i],num_elements_per_chunk*sizeof(double),hipMemcpyDeviceToHost,stream_copydh);
	}

	/*
	//Copy data to device
	checkCudaErrors(hipMemcpy(d_data,h_data,total_data*sizeof(double),hipMemcpyHostToDevice));

	//Figure out how many blocks are needed
	int blocksize = 256;
	int nblocks = (total_data-1)/blocksize+1;

	//Launch kernel to process data
	do_work<<<nblocks,blocksize,0,0>>>(d_data, total_data, 0*nblocks);

	//Copy data back from device
	checkCudaErrors(hipMemcpy(h_data,d_data,total_data*sizeof(double),hipMemcpyDeviceToHost));
	*/
	hipDeviceSynchronize();
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&time, start, stop));
	printf("Total processing time:  %g ms\n", time);

	//checkCudaErrors(hipFree( d_data ));
	free(h_data);
	for (int i = 0; i < num_chunks; ++i) 
	{
		checkCudaErrors(hipHostFree(h_data_list[i]));
		checkCudaErrors(hipFree(d_data_list[i]));
	}
	hipDeviceReset();
	return EXIT_SUCCESS;
}

